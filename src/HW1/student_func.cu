#include "hip/hip_runtime.h"
#include "utils.h"
#include ""



const size_t blockWidth = 32; //threads per block on one dimension (32*32 total)



__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
unsigned char* const greyImage,
size_t numRows, size_t numCols)
{
//Fill in the kernel to convert from color to greyscale
//the mapping from components of a uchar4 to RGBA is:
// .x -> R ; .y -> G ; .z -> B ; .w -> A
//
//The output (greyImage) at each pixel should be the result of
//applying the formula: output = .299f * R + .587f * G + .114f * B;
//Note: We will be ignoring the alpha channel for this conversion



//First create a mapping from the 2D block and grid locations
//to an absolute 2D location in the image, then use that to
//calculate a 1D offset
size_t idx_x = threadIdx.x + blockIdx.x*blockDim.x;
size_t idx_y = threadIdx.y + blockIdx.y*blockDim.y;



if (idx_x >= numRows || idx_y >= numCols) return; //it can happen on the "remainder" block

size_t idxvec = idx_x*numCols + idx_y;
uchar4 rgb_value = rgbaImage[idxvec];
greyImage[idxvec] = (unsigned char)(.299f*rgb_value.x + .587f*rgb_value.y + .114f*rgb_value.z);
}



void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
//You must fill in the correct sizes for the blockSize and gridSize
//currently only one block with one thread is being launched

const dim3 blockSize(blockWidth,blockWidth, 1);
unsigned int numBlocksX = (unsigned int)(numRows / blockWidth + 1);
unsigned int numBlocksY = (unsigned int)(numCols / blockWidth + 1);
const dim3 gridSize(numBlocksX,numBlocksY, 1);
rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);

hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());



}
